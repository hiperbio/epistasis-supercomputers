#include "hip/hip_runtime.h"
/**
 *
 * mpi-Epi4Tensor: Multi-GPU and multi-node epistasis detection specialized to fourth-order searches on modern GPU microarchitectures with matrix processing cores
 *
 * High-Performance Computing Architectures and Systems (HPCAS) Group, INESC-ID
 * Contact: Ricardo Nobre <ricardo.nobre@inesc-id.pt>
 *
 */

/* Standard Library */
#include <iostream>
#include <iomanip>      
#include <sstream>
#include <vector>
#include <cfloat>
#include <string>
#include <libgen.h>	
#include <mpi.h>

#include "cutlass-op.hpp"
#include "epistasis.hpp"
#include "reduction.hpp"

#define MAX_CHAR_ARRAY 1000

#define NUM_GPUS 1

/* Used for allocating arrays of binary data */
typedef typename cutlass::Array<cutlass::uint1b_t, 32> ScalarBinary32;


/* Reads number of SNPs, number of controls/cases and controls/cases file name. */
int readDatasetDescriptionFile(const char *fileName, uint *numSNPs, char *controlsFileName, uint *numControls, char *casesFileName, uint *numCases) {

	FILE *fStream = fopen(fileName, "r");                	// File with information and pointers to dataset.
	if(fStream == NULL) {
		std::cerr << "File '" << fileName << "' does not exist!" << std::endl;
		return 1;
	}

	char line[MAX_CHAR_ARRAY];
	char *ret = fgets(line, MAX_CHAR_ARRAY, fStream);       // First line represents the number of SNPs.
	*numSNPs = atoi(line);

	ret = fgets(controlsFileName, MAX_CHAR_ARRAY, fStream); // Second line represents the filename with controls data.
	controlsFileName[strcspn(controlsFileName, "\n")] = 0;  // Removes trailing newline character.

	ret = fgets(line, MAX_CHAR_ARRAY, fStream);             // Third line represents the number of controls.
	*numControls = atoi(line);

	ret = fgets(casesFileName, MAX_CHAR_ARRAY, fStream);    // Forth line represents the filename with cases data.
	casesFileName[strcspn(casesFileName, "\n")] = 0;        // Removes trailing newline character.

	ret = fgets(line, MAX_CHAR_ARRAY, fStream);             // Fifth line represents the number of cases.
	*numCases = atoi(line);

	return 0;
}

/* Reads genotypic data from samples of a given kind (cases or controls). */
int readDatasetSamplesData(const char *fileName, uint *datasetSamplesPacked32, uint numSNPs, uint numSamplesPacked32) {

	size_t numElem;
	FILE *ifpSamples = fopen(fileName, "rb");

	uint numElemToRead = numSamplesPacked32 * numSNPs * SNP_CALC;

	numElem = fread(datasetSamplesPacked32, sizeof(unsigned int), numSamplesPacked32 * numSNPs * SNP_CALC, ifpSamples);

	if(numElem != numElemToRead) {
		std::cerr << "Problem loading samples from storage device" << std::endl;
		return 1;
	}

	fclose(ifpSamples);	
	return 0;
}

/* Calculates nCk (number of choices of 'k' items from 'n' items), i.e.  --->  n! / (k!(n-k)!)
   Used to calculate the achieved number of SNP combinations evaluated per second.
 */
unsigned long long n_choose_k(unsigned int n, unsigned int k)
{
	unsigned long long result = 1;		// nC0

	for (unsigned int i = 1; i <= k; i++) {	// nC1 until nCk
		result = result * n / i;	// calculates nC_{i} from nC_{i-1}
	n = n - 1;
	}

	return result;
}

/* Performs a fourth-order epistasis detection search on one of more GPUs.
 */
hipError_t epistasisDetectionSearch(unsigned int* datasetCases_hostMatrixA, unsigned int* datasetControls_hostMatrixA, int numSNPs, int numCases, int numControls, uint numSNPsWithPadding, int numCasesWithPadding, int numControlsWithPadding, int * roundsCounter, double * searchTime, float * outputScore, unsigned long long int * outputIndices) {

	hipError_t result;

	/* Starts measuring time */
	struct timespec t_start, t_end;
	clock_gettime(CLOCK_MONOTONIC, &t_start);       


	/* Scores and SNP combination indices for different GPUs. NUM_GPUS is set in the Makefile */
	float outputScore_omp_arr[NUM_GPUS];	
	unsigned long long int outputIndices_omp_arr[NUM_GPUS];


	/* Constructs lgamma() lookup table. Used for calculation of K2 Bayesian scores */
	int tablePrecalc_size = max(numCases, numControls) + 1;
	float * h_tablePrecalc = (float*) malloc(tablePrecalc_size * sizeof(float));
	for(int i=1; i < (tablePrecalc_size + 1); i++) {
		h_tablePrecalc[i - 1] = lgamma((double)i);
	}


	(*roundsCounter) = 0;   

	double tensorTeraOperationsAcc = 0;


	/* Some parts of the code can be further simplified provided each MPI process is only assigned one GPU.
	 * This contrasts with the parallelization scheme that was being used before, which relied on OpenMP to facilitate targeting multiple GPUs on a single node. */
	
	// #pragma omp parallel num_threads(NUM_GPUS) reduction(+: tensorTeraOperationsAcc)
	// {

	int omp_thread_id = 0;	// omp_get_thread_num();
	hipSetDevice(omp_thread_id);


	/* GPU memory allocation for cases */

	ScalarBinary32 *cases_A_ptrGPU;
	ScalarBinary32 *cases_B_ptrGPU;
	result = hipMalloc((ScalarBinary32 **) &cases_A_ptrGPU, sizeof(ScalarBinary32) * numSNPsWithPadding * (numCasesWithPadding / 32) * SNP_CALC);		
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for cases input data." << std::endl;
	}


	result = hipMemcpyAsync(cases_A_ptrGPU, datasetCases_hostMatrixA, sizeof(int) * numSNPsWithPadding * (numCasesWithPadding / 32) * SNP_CALC, hipMemcpyHostToDevice, 0);

	cases_B_ptrGPU = cases_A_ptrGPU;	/* Makes matrix B points to the same data as matrix A */

	int *C_ptrGPU_cases;
	result = hipMalloc((int**) &C_ptrGPU_cases, sizeof(int) * NUM_STREAMS * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC));        

	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for cases output data." << std::endl;
	}


	/* GPU memory allocation for controls */

	ScalarBinary32 *controls_A_ptrGPU;
	ScalarBinary32 *controls_B_ptrGPU;
	result = hipMalloc((ScalarBinary32 **) &controls_A_ptrGPU, sizeof(ScalarBinary32) * numSNPsWithPadding * (numControlsWithPadding / 32) * SNP_CALC);	
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for controls input data." << std::endl;
	}

	result = hipMemcpyAsync(controls_A_ptrGPU, datasetControls_hostMatrixA, sizeof(int) * numSNPsWithPadding * (numControlsWithPadding / 32) * SNP_CALC, hipMemcpyHostToDevice, 0);

	controls_B_ptrGPU = controls_A_ptrGPU;	/* Makes matrix B points to the same data as matrix A */

	int *C_ptrGPU_controls;
	result = hipMalloc((int**) &C_ptrGPU_controls, sizeof(int) * NUM_STREAMS * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC));          

	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for controls output data." << std::endl;
	}


	/* Copies K2 score lookup table to (each) GPU device */

	float * d_tablePrecalc;
	result = hipMalloc((float**)&d_tablePrecalc, tablePrecalc_size * sizeof(float));
	result = hipMemcpy(d_tablePrecalc, h_tablePrecalc, tablePrecalc_size * sizeof(float), hipMemcpyHostToDevice);


	/* Allocates and initializes memory related to storing best score and indexes of corresponding set of SNPs */

	float * d_output;
	unsigned long long int * d_output_packedIndices;
	float h_output[1] = {FLT_MAX};
	result = hipMalloc((float**)&d_output, 1 * sizeof(float));								
	result = hipMalloc((unsigned long long int**)&d_output_packedIndices, 1 * sizeof(unsigned long long int));		
	result = hipMemcpy(d_output, h_output, 1 * sizeof(float), hipMemcpyHostToDevice);


	/* Setup of matrix-matrix operations using CUTLASS 2.X (tested on v2.5). */ 

	uint A_leadingDim_cases = numCasesWithPadding;             
	uint B_leadingDim_cases = numCasesWithPadding;             

	uint A_leadingDim_controls = numControlsWithPadding;       
	uint B_leadingDim_controls = numControlsWithPadding;       

	uint C_leadingDim = SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC;	


	/* Constructs contingency tables for individual SNPs */

	uint * d_output_individualSNP_popcountsForCases;
	uint * d_output_individualSNP_popcountsForControls;

	int blocksPerGrid_ind = (size_t)ceil(((float)(numSNPs)) / ((float)32));
	result = hipMalloc((uint**)&d_output_individualSNP_popcountsForControls, 3 * numSNPs * sizeof(uint));
	result = hipMalloc((uint**)&d_output_individualSNP_popcountsForCases, 3 * numSNPs * sizeof(uint));

	individualPopcount<<<blocksPerGrid_ind, 32, 0, 0>>>(0, (uint*)cases_A_ptrGPU, (uint*)controls_A_ptrGPU, d_output_individualSNP_popcountsForCases, d_output_individualSNP_popcountsForControls, numSNPs, numCases, numControls);  


	/* Constructs contingency tables for pairwise interactions of SNPs */

	uint * d_output_pairwiseSNP_popcountsForCases;
	uint * d_output_pairwiseSNP_popcountsForControls;
	result = hipMalloc((uint**)&d_output_pairwiseSNP_popcountsForControls, 9 * numSNPs * numSNPs * sizeof(uint));
	result = hipMalloc((uint**)&d_output_pairwiseSNP_popcountsForCases, 9 * numSNPs * numSNPs * sizeof(uint));

	dim3 blocksPerGrid_pairwise ( (size_t)ceil(((float)(numSNPs)) / ((float)16)), (size_t)ceil(((float)(numSNPs)) / ((float)16)), 1 );
	dim3 workgroupSize_pairwise ( 16, 16, 1 );

	pairwisePopcount<<<blocksPerGrid_pairwise, workgroupSize_pairwise, 0, 0>>>((uint*)cases_A_ptrGPU, (uint*)controls_A_ptrGPU, d_output_pairwiseSNP_popcountsForCases, d_output_pairwiseSNP_popcountsForControls, numSNPs, numCases, numControls, 0);


	/* Allocates space for pairwise combination of SNPs in CUDA cores.
	   Y_Z uses more space (x NUM_STREAMS) if using multiple streams for enabling concurrent execution of inner-loop (Z) iterations. */

	uint * d_output_WX_cases;
	uint * d_output_WX_controls;
	result = hipMalloc((uint**)&d_output_WX_cases, (numCasesWithPadding / 32) * (SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK) * sizeof(uint));		
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for cases pairwise popcounts." << std::endl;
	}
	result = hipMalloc((uint**)&d_output_WX_controls, (numControlsWithPadding / 32) * (SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK) * sizeof(uint));	
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for controls pairwise popcounts." << std::endl;
	}

	uint * d_output_YZ_cases;
	uint * d_output_YZ_controls;
	result = hipMalloc((uint**)&d_output_YZ_cases, NUM_STREAMS * (numCasesWithPadding / 32) * (SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK) * sizeof(uint));
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for cases pairwise popcounts." << std::endl;
	}
	result = hipMalloc((uint**)&d_output_YZ_controls, NUM_STREAMS * (numControlsWithPadding / 32) * (SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK) * sizeof(uint));
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for controls pairwise popcounts." << std::endl;
	}

	uint * d_output_XY_cases;
	uint * d_output_XY_controls;
	result = hipMalloc((uint**)&d_output_XY_cases, (numCasesWithPadding / 32) * (SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK) * sizeof(uint));
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for cases pairwise popcounts." << std::endl;
	}
	result = hipMalloc((uint**)&d_output_XY_controls, (numControlsWithPadding / 32) * (SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK) * sizeof(uint));
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for controls pairwise popcounts." << std::endl;
	}

	uint * d_output_WY_cases;
	uint * d_output_WY_controls;
	result = hipMalloc((uint**)&d_output_WY_cases, (numCasesWithPadding / 32) * (SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK) * sizeof(uint));
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for cases pairwise popcounts." << std::endl;
	}
	result = hipMalloc((uint**)&d_output_WY_controls, (numControlsWithPadding / 32) * (SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK) * sizeof(uint));
	if(result != hipSuccess) {
		std::cerr << "Failed allocating memory for controls pairwise popcounts." << std::endl;
	}



	/* Allocate memory for 3-way popcounts (calculated on tensor cores) */

	int *C_ptrGPU_cases_XYZ;
	result = hipMalloc((int**) &C_ptrGPU_cases_XYZ, sizeof(int) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (numSNPs * SNP_CALC));
	int *C_ptrGPU_controls_XYZ;
	result = hipMalloc((int**) &C_ptrGPU_controls_XYZ, sizeof(int) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (numSNPs * SNP_CALC));

	int *C_ptrGPU_cases_WYZ;
	result = hipMalloc((int**) &C_ptrGPU_cases_WYZ, sizeof(int) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (numSNPs * SNP_CALC));
	int *C_ptrGPU_controls_WYZ;
	result = hipMalloc((int**) &C_ptrGPU_controls_WYZ, sizeof(int) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (numSNPs * SNP_CALC));

	int *C_ptrGPU_cases_WXY;
	result = hipMalloc((int**) &C_ptrGPU_cases_WXY, sizeof(int) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (numSNPs * SNP_CALC));
	int *C_ptrGPU_controls_WXY;
	result = hipMalloc((int**) &C_ptrGPU_controls_WXY, sizeof(int) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (numSNPs * SNP_CALC));



	/* CUDA stream creation */

	hipStream_t cudaStream_prework_k3_WX, cudaStream_prework_k3_XY, cudaStream_prework_k3_WY;	
	hipStreamCreate(&cudaStream_prework_k3_WX);
	hipStreamCreate(&cudaStream_prework_k3_XY);
	hipStreamCreate(&cudaStream_prework_k3_WY);

	hipStream_t cudaStreamToUse[NUM_STREAMS];
	for (int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&cudaStreamToUse[i]);
	}


	/* Main loop performing the SNP evaluation rounds. */

	MPI_Status stat;
	int start_W;
	int start_X;

	unsigned long long start_W_and_X;

	uint objectiveFunctionIndex = 0;

	// #pragma omp for schedule(TYPE_SCHEDULING)
	// for(int start_W = 0; start_W < numSNPsWithPadding; start_W+=SNP_BLOCK) {
	while (1) {

		MPI_Send (NULL, 0 , MPI_INT, 0 /* goes to rank 0 */, 0 /* only one type of tag */, MPI_COMM_WORLD);	

		// Gets data from master
		MPI_Recv (&start_W_and_X , 1, MPI_UNSIGNED_LONG_LONG, 0 /* comes from rank 0 */, 0 /* only one type of tag */, MPI_COMM_WORLD, &stat);

		if (start_W_and_X == 0xffffffffffffffff) {	
			break;
		}

		start_W = start_W_and_X & 0xFFFFFFFF;
		start_X = (start_W_and_X >> 32) & 0xFFFFFFFF; 

		std::cout << "Loop iteration W=" << (int) (start_W / SNP_BLOCK) << " and X=" << (int) (start_X / SNP_BLOCK) << " out of " << (int) (numSNPsWithPadding / SNP_BLOCK) << std::endl;	

		/* In case the last calls to the 'applyScore_and_FindGloballyBestSol()' GPU kernel did not terminate yet at this point. */
		for(int i=0; i<NUM_STREAMS; i++) {
			hipStreamSynchronize(cudaStreamToUse[i]);
		}


		/* Combines an SNP W with a block of SNPs X. */

		dim3 blocksPerGrid_prework_k3_WX( (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), 1);
		dim3 workgroupSize_prework_k3_WX( 1, 1, 64 );     
		combine<<<blocksPerGrid_prework_k3_WX, workgroupSize_prework_k3_WX, 0, cudaStream_prework_k3_WX >>>(((uint*)cases_A_ptrGPU), ((uint*)controls_A_ptrGPU), d_output_WX_cases, d_output_WX_controls, numSNPs, numCases, numControls, start_W, start_X); 

		/* ** TENSOR ** Constructs contingency tables for W_X_Y (also used as W_X_Z) */

		// Processes Cases
		int SNP_Y_index_start = ((int)(start_X / 64.0)) * 64;	
		ScalarBinary32 *A_ptrGPU_iter_cases_WXY = (ScalarBinary32 *) (d_output_WX_cases);   
		ScalarBinary32 *B_ptrGPU_iter_cases_WXY = cases_B_ptrGPU  +  (SNP_Y_index_start * SNP_CALC * (numCasesWithPadding/32));	


		result = cutlass_U1_MmaMatOpTN(
				(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),
				(numSNPsWithPadding - SNP_Y_index_start) * (SNP_CALC),
				numCasesWithPadding,
				(cutlass::uint1b_t*) A_ptrGPU_iter_cases_WXY,
				A_leadingDim_cases,
				(cutlass::uint1b_t*) B_ptrGPU_iter_cases_WXY,
				B_leadingDim_cases,
				C_ptrGPU_cases_WXY + (SNP_Y_index_start * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC * SNP_CALC))),	
				C_leadingDim,   
				cudaStream_prework_k3_WX 
				);

		if(result != hipSuccess) {
			std::cout << "Problem in construction of contingency tables for W_X_Y (cases)." << std::endl;
		}

		tensorTeraOperationsAcc += (double) numCasesWithPadding * (double)(SNP_BLOCK * SNP_BLOCK) * (double)(SNP_CALC * SNP_CALC) * (double)(numSNPsWithPadding - SNP_Y_index_start) * (double)(SNP_CALC);


		// Processes Controls

		ScalarBinary32 *A_ptrGPU_iter_controls_WXY = (ScalarBinary32 *) (d_output_WX_controls);     
		ScalarBinary32 *B_ptrGPU_iter_controls_WXY = controls_B_ptrGPU  +  (SNP_Y_index_start * SNP_CALC * (numControlsWithPadding/32));

		result = cutlass_U1_MmaMatOpTN(
				(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),
				(numSNPsWithPadding - SNP_Y_index_start) * (SNP_CALC),
				numControlsWithPadding,
				(cutlass::uint1b_t*)A_ptrGPU_iter_controls_WXY,
				A_leadingDim_controls,
				(cutlass::uint1b_t*) B_ptrGPU_iter_controls_WXY,
				B_leadingDim_controls,
				C_ptrGPU_controls_WXY	+       (SNP_Y_index_start * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC * SNP_CALC))),
				C_leadingDim,   
				cudaStream_prework_k3_WX	
				);

		if(result != hipSuccess) {
			std::cout << "Problem in construction of contingency tables for W_X_Y (controls)." << std::endl;
		}

		tensorTeraOperationsAcc += (double) numControlsWithPadding * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC) * (double) (numSNPsWithPadding - SNP_Y_index_start) * (double) (SNP_CALC);



		for(int start_Y = start_X; start_Y < numSNPsWithPadding; start_Y+=SNP_BLOCK) {

			/* In case the last calls to the 'applyScore_and_FindGloballyBestSol()' GPU kernel did not terminate yet at this point. */
			for(int i=0; i<NUM_STREAMS; i++) {
				hipStreamSynchronize(cudaStreamToUse[i]);
			}


			/* Combines a block of SNPs X with a block of SNPs Y. Used for the construction of contingency tables for 3rd order SNP interactions. */

			dim3 blocksPerGrid_prework_k3_XY( (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), 1);
			dim3 workgroupSize_prework_k3_XY( 1, 1, 64 );     
			combine<<<blocksPerGrid_prework_k3_XY, workgroupSize_prework_k3_XY, 0, cudaStream_prework_k3_XY >>>(((uint*)cases_A_ptrGPU), ((uint*)controls_A_ptrGPU), d_output_XY_cases, d_output_XY_controls, numSNPs, numCases, numControls, start_X, start_Y);


			/* Combines a block of SNPs W with a block of SNPs Y. Used for the construction of contingency tables for 3rd order SNP interactions. */

			dim3 blocksPerGrid_prework_k3_WY( (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), 1);
			dim3 workgroupSize_prework_k3_WY( 1, 1, 64 );     
			combine<<<blocksPerGrid_prework_k3_WY, workgroupSize_prework_k3_WY, 0, cudaStream_prework_k3_WY >>>(((uint*)cases_A_ptrGPU), ((uint*)controls_A_ptrGPU), d_output_WY_cases, d_output_WY_controls, numSNPs, numCases, numControls, start_W, start_Y);	


			/* ** TENSOR ** Constructs contingency tables for W_Y_Z */

			// Processes Cases

			int SNP_Z_index_start = ((int)(start_Y / 64.0)) * 64;	
			ScalarBinary32 *A_ptrGPU_iter_cases_WYZ = (ScalarBinary32 *) (d_output_WY_cases);   
			ScalarBinary32 *B_ptrGPU_iter_cases_WYZ = cases_B_ptrGPU +  (SNP_Z_index_start * SNP_CALC * (numCasesWithPadding/32));        

			result = cutlass_U1_MmaMatOpTN(
					(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),
					(numSNPsWithPadding - SNP_Z_index_start) * (SNP_CALC),
					numCasesWithPadding,
					(cutlass::uint1b_t*) A_ptrGPU_iter_cases_WYZ,
					A_leadingDim_cases,
					(cutlass::uint1b_t*) B_ptrGPU_iter_cases_WYZ,
					B_leadingDim_cases,
					C_ptrGPU_cases_WYZ +	(SNP_Z_index_start * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC * SNP_CALC))),
					C_leadingDim,   
					cudaStream_prework_k3_WY	
					);

			if(result != hipSuccess) {
				std::cout << "Problem in construction of contingency tables for W_Y_Z (cases)." << std::endl;
			}


			tensorTeraOperationsAcc += (double) numCasesWithPadding * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC) * (double) (numSNPsWithPadding - SNP_Z_index_start) * (double) (SNP_CALC);


			// Processes Controls

			ScalarBinary32 *A_ptrGPU_iter_controls_WYZ = (ScalarBinary32 *) (d_output_WY_controls);     
			ScalarBinary32 *B_ptrGPU_iter_controls_WYZ = controls_B_ptrGPU +  (SNP_Z_index_start * SNP_CALC * (numControlsWithPadding/32));

			result = cutlass_U1_MmaMatOpTN(
					(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),
					(numSNPsWithPadding - SNP_Z_index_start) * (SNP_CALC),
					numControlsWithPadding,
					(cutlass::uint1b_t*)A_ptrGPU_iter_controls_WYZ,
					A_leadingDim_controls,
					(cutlass::uint1b_t*) B_ptrGPU_iter_controls_WYZ,
					B_leadingDim_controls,
					C_ptrGPU_controls_WYZ +	(SNP_Z_index_start * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC * SNP_CALC))),
					C_leadingDim,   
					cudaStream_prework_k3_WY	
					);

			if(result != hipSuccess) {
				std::cout << "Problem in construction of contingency tables for W_Y_Z (controls)." << std::endl;
			}

			tensorTeraOperationsAcc += (double) numControlsWithPadding * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC) * (double) (numSNPsWithPadding - SNP_Z_index_start) * (double) (SNP_CALC);


			/* ** TENSOR ** Constructs contingency tables for X_Y_Z */

			// Processes Cases

			ScalarBinary32 *A_ptrGPU_iter_cases_XYZ = (ScalarBinary32 *) (d_output_XY_cases);   
			ScalarBinary32 *B_ptrGPU_iter_cases_XYZ = cases_B_ptrGPU +  (SNP_Z_index_start * SNP_CALC * (numCasesWithPadding/32));        

			result = cutlass_U1_MmaMatOpTN(
					(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),
					(numSNPsWithPadding - SNP_Z_index_start) * (SNP_CALC),
					numCasesWithPadding,
					(cutlass::uint1b_t*) A_ptrGPU_iter_cases_XYZ,
					A_leadingDim_cases,
					(cutlass::uint1b_t*) B_ptrGPU_iter_cases_XYZ,
					B_leadingDim_cases,
					C_ptrGPU_cases_XYZ +       (SNP_Z_index_start * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC * SNP_CALC))),
					C_leadingDim,   
					cudaStream_prework_k3_XY	
					);

			if(result != hipSuccess) {
				std::cout << "Problem in construction of contingency tables for X_Y_Z (cases)." << std::endl;
			}


			tensorTeraOperationsAcc += (double) numCasesWithPadding * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC) * (double) (numSNPsWithPadding - SNP_Z_index_start) * (double) (SNP_CALC);


			// Processes Controls

			ScalarBinary32 *A_ptrGPU_iter_controls_XYZ = (ScalarBinary32 *) (d_output_XY_controls);     
			ScalarBinary32 *B_ptrGPU_iter_controls_XYZ = controls_B_ptrGPU +  (SNP_Z_index_start * SNP_CALC * (numControlsWithPadding/32));

			result = cutlass_U1_MmaMatOpTN(
					(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),
					(numSNPsWithPadding - SNP_Z_index_start) * (SNP_CALC),
					numControlsWithPadding,
					(cutlass::uint1b_t*)A_ptrGPU_iter_controls_XYZ,
					A_leadingDim_controls,
					(cutlass::uint1b_t*) B_ptrGPU_iter_controls_XYZ,
					B_leadingDim_controls,
					C_ptrGPU_controls_XYZ +	(SNP_Z_index_start * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC * SNP_CALC))),
					C_leadingDim,   
					cudaStream_prework_k3_XY 
					);

			if(result != hipSuccess) {
				std::cout << "Problem in construction of contingency tables for X_Y_Z (controls)." << std::endl;
			}

			tensorTeraOperationsAcc += (double) numControlsWithPadding * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC) * (double) (numSNPsWithPadding - SNP_Z_index_start) * (double) (SNP_CALC);


			for(int start_Z = start_Y; start_Z < numSNPsWithPadding; start_Z+=SNP_BLOCK) {

				// #pragma omp atomic	
				(*roundsCounter)++;


				/* Combines a block of SNPs Y with a block of SNPs Z. */

				dim3 blocksPerGrid_prework_k3_YZ( (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), 1);
				dim3 workgroupSize_prework_k3_YZ( 1, 1, 64 );     
				combine<<<blocksPerGrid_prework_k3_YZ, workgroupSize_prework_k3_YZ, 0, cudaStreamToUse[(objectiveFunctionIndex % NUM_STREAMS)] >>>(((uint*)cases_A_ptrGPU), ((uint*)controls_A_ptrGPU), d_output_YZ_cases + (objectiveFunctionIndex % NUM_STREAMS) * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC) * (numCasesWithPadding / 32)), d_output_YZ_controls + (objectiveFunctionIndex % NUM_STREAMS) * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC) * (numControlsWithPadding / 32)), numSNPs, numCases, numControls, start_Y, start_Z);     


				if((start_Y == start_X) && (start_Z == start_Y) ) {	
					hipStreamSynchronize(cudaStream_prework_k3_WX);	
				}



				/* Main calculation in 4-way (using tensor cores). 
				   Combines block of (SNP_BLOCK * SNP_BLOCK) 2-way pairings (WX) with (SNP_BLOCK * SNP_BLOCK) 2-way pairings (YZ) of SNPs.
				 */


				// Processes Cases

				ScalarBinary32 *A_ptrGPU_iter_cases = (ScalarBinary32 *) (d_output_WX_cases);	
				ScalarBinary32 *B_ptrGPU_iter_cases = (ScalarBinary32 *) (d_output_YZ_cases) + (objectiveFunctionIndex % NUM_STREAMS) * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC) * (numCasesWithPadding / 32)); 

				result = cutlass_U1_MmaMatOpTN(
						(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),   
						(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),       
						numCasesWithPadding,            
						(cutlass::uint1b_t*) A_ptrGPU_iter_cases,
						A_leadingDim_cases,
						(cutlass::uint1b_t*) B_ptrGPU_iter_cases,
						B_leadingDim_cases,
						C_ptrGPU_cases + (objectiveFunctionIndex % NUM_STREAMS) * (((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC)) * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC))),  
						C_leadingDim,
						cudaStreamToUse[(objectiveFunctionIndex % NUM_STREAMS)] 
						);

				if(result != hipSuccess) {
					std::cout << "Problem in construction of contingency tables for W_X_Y_Z (cases)." << std::endl;
				}

				tensorTeraOperationsAcc += (double) numCasesWithPadding * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC) * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC);


				// Processes Controls

				ScalarBinary32 *A_ptrGPU_iter_controls = (ScalarBinary32 *) (d_output_WX_controls);	
				ScalarBinary32 *B_ptrGPU_iter_controls = (ScalarBinary32 *) (d_output_YZ_controls) + (objectiveFunctionIndex % NUM_STREAMS) * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC) * (numControlsWithPadding / 32));

				result = cutlass_U1_MmaMatOpTN(
						(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),
						(SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC),
						numControlsWithPadding,         
						(cutlass::uint1b_t*)A_ptrGPU_iter_controls,
						A_leadingDim_controls,
						(cutlass::uint1b_t*) B_ptrGPU_iter_controls,
						B_leadingDim_controls,
						C_ptrGPU_controls + (objectiveFunctionIndex % NUM_STREAMS) * (((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC)) * ((SNP_BLOCK * SNP_BLOCK) * (SNP_CALC * SNP_CALC))),
						C_leadingDim,
						cudaStreamToUse[(objectiveFunctionIndex % NUM_STREAMS)] 
						);

				if(result != hipSuccess) {
					std::cout << "Problem in construction of contingency tables for W_X_Y_Z (controls)." << std::endl;
				}

				tensorTeraOperationsAcc += numControlsWithPadding * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC) * (double) (SNP_BLOCK * SNP_BLOCK) * (double) (SNP_CALC * SNP_CALC);


				if(start_Z == start_Y) {
					hipStreamSynchronize(cudaStream_prework_k3_WY);    
					hipStreamSynchronize(cudaStream_prework_k3_XY);
				}


				/* Call K2 objective scoring function */

				dim3 blocksPerGrid_objFun( (size_t)ceil(((float)(SNP_BLOCK)) / ((float)SNP_BLOCK) ), (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)), (size_t)ceil(((float)(SNP_BLOCK)) / ((float)1)));		
				dim3 workgroupSize_objFun( SNP_BLOCK, 1, 1 );	

				applyScore_and_FindGloballyBestSol<<<blocksPerGrid_objFun, workgroupSize_objFun, 0, cudaStreamToUse[(objectiveFunctionIndex % NUM_STREAMS)]>>>(C_ptrGPU_cases + (objectiveFunctionIndex % NUM_STREAMS) * ((SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC)), C_ptrGPU_controls + (objectiveFunctionIndex % NUM_STREAMS) * ((SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC) * (SNP_BLOCK * SNP_BLOCK * SNP_CALC * SNP_CALC)), C_ptrGPU_cases_XYZ, C_ptrGPU_controls_XYZ, C_ptrGPU_cases_WYZ, C_ptrGPU_controls_WYZ, C_ptrGPU_cases_WXY, C_ptrGPU_controls_WXY, d_output_individualSNP_popcountsForCases, d_output_individualSNP_popcountsForControls, d_output_pairwiseSNP_popcountsForCases, d_output_pairwiseSNP_popcountsForControls, d_tablePrecalc, d_output, d_output_packedIndices, start_W, start_X, start_Y, start_Z, numSNPs, numCases, numControls);


				objectiveFunctionIndex = (objectiveFunctionIndex + 1) % NUM_STREAMS;	

			}
		}
	}



	/* In case evaluation rounds are still executing */
	for (int i = 0; i < NUM_STREAMS; i++) {
		hipStreamSynchronize(cudaStreamToUse[i]);
	}

	/* Copies best solution found from GPU memory to Host */
	hipMemcpy(&outputScore_omp_arr[omp_thread_id], d_output, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&outputIndices_omp_arr[omp_thread_id], d_output_packedIndices, sizeof(unsigned long long int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	hipFree(cases_A_ptrGPU);
	hipFree(C_ptrGPU_cases);
	hipFree(controls_A_ptrGPU);
	hipFree(C_ptrGPU_controls);
	hipFree(d_output_individualSNP_popcountsForControls);
	hipFree(d_output_individualSNP_popcountsForCases);
	hipFree(d_tablePrecalc);
	hipFree(d_output);
	hipFree(d_output_packedIndices);

	// }	// closes 'pragma omp parallel'


	*outputScore = FLT_MAX;
	for(int i=0; i<NUM_GPUS; i++) {
		if(outputScore_omp_arr[i] < (*outputScore)) {
			*outputScore = outputScore_omp_arr[i];
			*outputIndices = outputIndices_omp_arr[i];
		}
	}

	clock_gettime(CLOCK_MONOTONIC, &t_end); // final timestamp

	(*searchTime) = ((t_end.tv_sec + ((double) t_end.tv_nsec / 1000000000)) - (t_start.tv_sec + ((double) t_start.tv_nsec / 1000000000)));

	std::cout << "Tensor TOPS: " << std::fixed << std::setprecision(3) << ((double) (tensorTeraOperationsAcc / (*searchTime) * 2) / 1e12) << std::endl;	// '* 2' because each AND+POC (or XOR+POPC) counts as two operations

	free(h_tablePrecalc);

	return hipSuccess;
}


/* Entry point of the application. */
int main(int argc, const char *arg[]) {

	hipError_t result;

	uint numSNPs, numControls, numCases;
	char controlsFileName[MAX_CHAR_ARRAY], casesFileName[MAX_CHAR_ARRAY];

	if(argc < 2) {
		std::cerr << "Usage: epi4tensor dataset.txt" << std::endl;
		return 1;
	}

	char *ts = strdup(arg[1]);
	char *pathToDataset = dirname(ts);

	/* Reads information about input dataset (number of SNPs, controls and cases, and controls/cases file names) from description file. */
	readDatasetDescriptionFile(arg[1], &numSNPs, controlsFileName, &numControls, casesFileName, &numCases);

	/* Calculates number of distinct blocks and padds number of SNPs to process to the block size. */
	uint numBlocks = ceil((float)numSNPs / (float)SNP_BLOCK);
	uint numSNPsWithPadding = numBlocks * SNP_BLOCK;

	/* Padds the number of controls and of cases. */
	uint numCasesWithPadding = ceil((float)numCases / PADDING_SAMPLES) * PADDING_SAMPLES;	
	uint numControlsWithPadding = ceil((float)numControls / PADDING_SAMPLES) * PADDING_SAMPLES;


	/* Prints information about dataset and number of distinct blocks of SNPs to process. */
	std::cout << "Num. SNPs: " << numSNPs << std::endl;
	std::cout << "Num. Cases: " << numCases << std::endl;
	std::cout << "Num. Controls: " << numControls << std::endl;


	/* Allocates pinned memory for holding controls and cases dataset matrices.
	   Each 32-bit 'unsigned int' holds 32 binary values representing genotype information.
	   Only two allele types are represented (SNP_CALC macro equals 2), ...
	   ... being information about the third allele type infered.
	 */


	int numCasesPacked32 = ceil(((float) numCasesWithPadding) / 32.0f);
	int numControlsPacked32 = ceil(((float) numControlsWithPadding) / 32.0f);

	int datasetCasesPacked32_size = numCasesPacked32 * numSNPsWithPadding * SNP_CALC;
	int datasetControlsPacked32_size = numControlsPacked32 * numSNPsWithPadding * SNP_CALC;

	unsigned int *datasetCasesPacked32 = NULL, *datasetControlsPacked32 = NULL;

	result = hipHostAlloc((void**)&datasetCasesPacked32, datasetCasesPacked32_size * sizeof(unsigned int), hipHostMallocDefault );     
	result = hipHostAlloc((void**)&datasetControlsPacked32, datasetControlsPacked32_size * sizeof(unsigned int), hipHostMallocDefault );
	if((datasetCasesPacked32 == NULL) || (datasetControlsPacked32 == NULL)) {
		std::cerr << "Problem allocating Host memory for cases and/or controls" << std::endl;
	}


	/* Reads dataset (controls and cases data) from storage device.
	   Input dataset must be padded with zeros in the dimension of samples (cases / controls), ...
	   ... making the number of bits per {SNP, allele} tuple a multiple of PADDING_SAMPLES. */

	std::string absolutePathToCasesFile = std::string(pathToDataset) + "/" + casesFileName;
	std::string absolutePathToControlsFile = std::string(pathToDataset) + "/" + controlsFileName;

	readDatasetSamplesData(absolutePathToCasesFile.c_str(), datasetCasesPacked32, numSNPs, numCasesPacked32);
	readDatasetSamplesData(absolutePathToControlsFile.c_str(), datasetControlsPacked32, numSNPs, numControlsPacked32);


	std::cout << "-------------------------------" << std::endl;




	/* Initializes the MPI environment */
	MPI_Init(NULL, NULL);

	/* Gets the number of processes */
	int mpi_world_size;
	MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

	/* Gets the rank of the process */
	int mpi_world_rank;
	MPI_Comm_rank(MPI_COMM_WORLD, &mpi_world_rank);


	/* Gets the name of the processor */
	char processor_name[MPI_MAX_PROCESSOR_NAME];
	int name_len;
	MPI_Get_processor_name(processor_name, &name_len);

	/* Prints a hello world message */
	printf("Hello world from processor %s, rank %d out of %d processors\n", processor_name, mpi_world_rank, mpi_world_size);

	if(mpi_world_rank == 0) {
		printf("Using %d MPI processes.\n", mpi_world_size);
	}

	MPI_Barrier(MPI_COMM_WORLD); /* All nodes are ready */

	struct timespec t_start, t_end;
	clock_gettime(CLOCK_MONOTONIC, &t_start);       // initial timestamp


	unsigned int start_W = 0;
	unsigned int start_X = 0;	

	unsigned long long start_W_and_X;

	int numActiveSlaves = mpi_world_size - 1;	

	float outputScore = FLT_MAX;
	unsigned long long int outputIndices;


	if (mpi_world_rank == 0) {

		MPI_Status mpiStatus;

		while ( numActiveSlaves > 0 ) {

			MPI_Recv(NULL, 0, MPI_INT, MPI_ANY_SOURCE, 0 /* only one type of tag */, MPI_COMM_WORLD, &mpiStatus);
			int slaveRank = mpiStatus.MPI_SOURCE;


			if (start_W < numSNPsWithPadding) {	

				start_W_and_X = (((unsigned long long int) start_W) << 0) | (((unsigned long long int) start_X) << 32);
				MPI_Send(&start_W_and_X, 1, MPI_UNSIGNED_LONG_LONG , slaveRank, 0 /* only one type of tag */, MPI_COMM_WORLD);

				start_X += SNP_BLOCK;

				if(start_X >= numSNPsWithPadding) {	
					start_W += SNP_BLOCK;
					start_X = start_W;
				}



			} else {

				unsigned long long stopSlave = 0xffffffffffffffff;	
				MPI_Send (&stopSlave, 1, MPI_UNSIGNED_LONG_LONG, slaveRank , 0 /* only one type of tag */ , MPI_COMM_WORLD);

				numActiveSlaves--;
			}
		}


	} else {

		/* Launches epistasis detection search. */

		int roundsCounter;
		double searchTime;

		result = epistasisDetectionSearch(
				datasetCasesPacked32,			// Cases matrix.
				datasetControlsPacked32,		// Controls matrix.
				numSNPs,                           	// Number of SNPs.
				numCases,                           	// Number of cases.
				numControls,                        	// Number of controls.
				numSNPsWithPadding,                 	// Number of SNPs padded to block size.
				numCasesWithPadding,     		// Number of cases padded to PADDING_SIZE.
				numControlsWithPadding,     		// Number of controls padded to PADDING_SIZE.
				&roundsCounter,			// Counter for number of rounds processed.
				&searchTime,				// Counter for execution time (seconds).
				&outputScore,				// Score of best score found.
				&outputIndices				// Indices of SNPs of set that results in best score.
				);

		if(result != 0) {
			std::cerr << "Epistasis detection search failed." << std::endl;
		}

	}


	/* Reduces scores and gets indices of best score */

	float * outputScorePerMpiProcess; 
	unsigned long long int * outputIndicesPerMpiProcess;

	if ( mpi_world_rank == 0) { 
		outputScorePerMpiProcess = (float *)malloc(mpi_world_size*1*sizeof(float)); 
		outputIndicesPerMpiProcess = (unsigned long long int *)malloc(mpi_world_size*1*sizeof(unsigned long long int));

	} 

	MPI_Gather( &outputScore, 1, MPI_FLOAT, outputScorePerMpiProcess, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);	// 0 is the root process
	MPI_Gather( &outputIndices, 1, MPI_UNSIGNED_LONG_LONG, outputIndicesPerMpiProcess, 1, MPI_UNSIGNED_LONG_LONG, 0, MPI_COMM_WORLD);

	float outputScoreAllMpiProcesses = FLT_MAX;
	unsigned long long outputIndicesAllMpiProcesses;

	if(mpi_world_rank == 0) {
		for(int i = 0; i < mpi_world_size; i++) {
			if(outputScorePerMpiProcess[i] < outputScoreAllMpiProcesses) {
				outputScoreAllMpiProcesses = outputScorePerMpiProcess[i];
				outputIndicesAllMpiProcesses = outputIndicesPerMpiProcess[i];
			}
		}

	}


	MPI_Barrier(MPI_COMM_WORLD);
	clock_gettime(CLOCK_MONOTONIC, &t_end);

	MPI_Finalize();


	if(mpi_world_rank == 0) {

		/* Prints indices of set of SNPs most associated with phenotype and its score */

		double timing_duration_mpi = ((t_end.tv_sec + ((double) t_end.tv_nsec / 1000000000)) - (t_start.tv_sec + ((double) t_start.tv_nsec / 1000000000)));

		std::cout << "-------------------------------" << std::endl << "{SNP_W_i, SNP_X_i, SNP_Y_i, SNP_Z_i}: SCORE\t->\t{" << ((outputIndicesAllMpiProcesses >> 0) & 0xFFFF) << ", " << ((outputIndicesAllMpiProcesses >> 16) & 0xFFFF) << ", " << ((outputIndicesAllMpiProcesses >> 32) & 0xFFFF) << ", " << ((outputIndicesAllMpiProcesses >> 48) & 0xFFFF) << "}: " << std::fixed << std::setprecision(6) << outputScoreAllMpiProcesses << std::endl;

		/* Prints time to execute the application, the achieved performance, and the ratio of unique sets */

		unsigned long long numCombinations = n_choose_k(numSNPs, INTER_OR);

		std::cout << "Wall-clock time:\t" << std::fixed << std::setprecision(3) << timing_duration_mpi << " seconds" << std::endl;    

		std::cout << "Num. unique sets per sec. (scaled to sample size): " << std::fixed << std::setprecision(3) << (((double) numCombinations * (double) (numCases + numControls) / (double)(timing_duration_mpi)) / 1e12) << " × 10^12" << std::endl;

		std::cout << "Unique sets of SNPs evaluated (k=" << INTER_OR << "): " << numCombinations << std::endl;

	}
	return result == hipSuccess ? 0 : 1;	

}





